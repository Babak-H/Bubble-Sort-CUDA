#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <ctime>
#include <limits>
#include <algorithm>
#include <Windows.h>

__global__ void SortGPU(int *a, int size)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x * 2;
	int cacheFirst;
	int cacheSecond;
	int cacheThird;

	for (int j = 0; j < size / 2 + 1; j++) 
	{

		if (i + 1 < size) 
		{
			cacheFirst = a[i];
			cacheSecond = a[i + 1];

			if (cacheFirst > cacheSecond) 
			{
				int temp = cacheFirst;
				a[i] = cacheSecond;
				cacheSecond = a[i + 1] = temp;
			}
		}

		if (i + 2 < size) 
		{
			cacheThird = a[i + 2];
			if (cacheSecond > cacheThird) {
				int temp = cacheSecond;
				a[i + 1] = cacheThird;
				a[i + 2] = temp;
			}
		}
		__syncthreads();
	}
}


using namespace std;

int main()
{
	//int A[6] = { 6,5,3,2,1,4 };
	//int n = sizeof(A) / sizeof(*A);
	const int count = 512;
	int *h_a = new int[count];

	for (int i = 0; i < count; i++)
	{
		h_a[i] = rand() % 10000;
	}

	int *d_a;
	hipMalloc(&d_a, sizeof(int)*count);
	hipMemcpy(d_a, h_a, sizeof(int)*count, hipMemcpyHostToDevice);

	SortGPU<<<1, 256>>>(d_a, count);
	
	hipMemcpy(h_a, d_a, sizeof(int)*count, hipMemcpyDeviceToHost);
	 

	hipFree(d_a);
	delete[] h_a;

	hipDeviceReset();
	return 0;
}


void SortCPU(int A[], int count)
{
	int k = 0, x, y;

	for (k = 0; k < count - 1; k++)
	{
		for (int i = 0; i < count - 1 - k; i++)
		{
			int flag = 0;
			if (A[i] > A[i + 1])
			{
				x = A[i];
				y = A[i + 1];

				A[i] = y;
				A[i + 1] = x;
				flag = 1;
			}

			if (flag == 0)
			{
				break;
			}
		}
	}

	for (int i = 0; i < 6; i++)
	{
		cout << A[i] << endl;
	}
	cout << endl;
}
